#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"
#include "mpichconf.h"
#include "mpl_base.h"

extern "C"{
#include "ch4_cuda_kernel_ops.h"

/* Cuda specific implementation */
#define generate_kernel_MPIR_LBAND(type_name_, c_type_)                                          \
    __global__ void kernel_##type_name_##_##MPIR_LBAND(c_type_ *a, c_type_ *b, int n)            \
    {                                                                                           \
        printf("_______________Inside LBAND Kernel____________________");                       \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = ((a[i]) & (b[i]));                                                           \
        }                                                                                       \
    }

generate_kernel_MPIR_LBAND(mpir_typename_int, int);
generate_kernel_MPIR_LBAND(mpir_typename_long, long);
generate_kernel_MPIR_LBAND(mpir_typename_short, short);
generate_kernel_MPIR_LBAND(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPIR_LBAND(mpir_typename_unsigned, unsigned);
generate_kernel_MPIR_LBAND(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPIR_LBAND(mpir_typename_long_long, long long);
generate_kernel_MPIR_LBAND(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPIR_LBAND(mpir_typename_signed_char, signed char);
generate_kernel_MPIR_LBAND(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPIR_LBAND(mpir_typename_int8_t, int8_t);
generate_kernel_MPIR_LBAND(mpir_typename_int16_t, int16_t);
generate_kernel_MPIR_LBAND(mpir_typename_int32_t, int32_t);
generate_kernel_MPIR_LBAND(mpir_typename_int64_t, int64_t);
generate_kernel_MPIR_LBAND(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPIR_LBAND(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPIR_LBAND(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPIR_LBAND(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPIR_LBAND(mpir_typename_char, char);
generate_kernel_MPIR_LBAND(mpir_typename_integer, MPI_Fint);
generate_kernel_MPIR_LBAND(mpir_typename_aint, MPI_Aint);
generate_kernel_MPIR_LBAND(mpir_typename_offset, MPI_Offset);
generate_kernel_MPIR_LBAND(mpir_typename_count, MPI_Count);
generate_kernel_MPIR_LBAND(mpir_typename_character, char);
generate_kernel_MPIR_LBAND(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPIR_LBAND(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPIR_LBAND(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPIR_LBAND(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPIR_LBAND(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPIR_LBAND(mpir_typename_byte, unsigned char);


#define generate_kernel_MPIR_LBOR(type_name_, c_type_)                                           \
    __global__ void kernel_##type_name_##_##MPIR_LBOR(c_type_ *a, c_type_ *b, int n)             \
    {                                                                                           \
        printf("_______________Inside LBOR Kernel____________________");                        \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = ((a[i]) | (b[i]));                                                           \
        }                                                                                       \
    }

generate_kernel_MPIR_LBOR(mpir_typename_int, int);
generate_kernel_MPIR_LBOR(mpir_typename_long, long);
generate_kernel_MPIR_LBOR(mpir_typename_short, short);
generate_kernel_MPIR_LBOR(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPIR_LBOR(mpir_typename_unsigned, unsigned);
generate_kernel_MPIR_LBOR(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPIR_LBOR(mpir_typename_long_long, long long);
generate_kernel_MPIR_LBOR(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPIR_LBOR(mpir_typename_signed_char, signed char);
generate_kernel_MPIR_LBOR(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPIR_LBOR(mpir_typename_int8_t, int8_t);
generate_kernel_MPIR_LBOR(mpir_typename_int16_t, int16_t);
generate_kernel_MPIR_LBOR(mpir_typename_int32_t, int32_t);
generate_kernel_MPIR_LBOR(mpir_typename_int64_t, int64_t);
generate_kernel_MPIR_LBOR(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPIR_LBOR(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPIR_LBOR(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPIR_LBOR(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPIR_LBOR(mpir_typename_char, char);
generate_kernel_MPIR_LBOR(mpir_typename_integer, MPI_Fint);
generate_kernel_MPIR_LBOR(mpir_typename_aint, MPI_Aint);
generate_kernel_MPIR_LBOR(mpir_typename_offset, MPI_Offset);
generate_kernel_MPIR_LBOR(mpir_typename_count, MPI_Count);
generate_kernel_MPIR_LBOR(mpir_typename_character, char);
generate_kernel_MPIR_LBOR(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPIR_LBOR(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPIR_LBOR(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPIR_LBOR(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPIR_LBOR(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPIR_LBOR(mpir_typename_byte, unsigned char);

#define generate_kernel_MPIR_LBXOR(type_name_, c_type_)                                          \
    __global__ void kernel_##type_name_##_##MPIR_LBXOR(c_type_ *a, c_type_ *b, int n)            \
    {                                                                                           \
        printf("_______________Inside LBXOR Kernel____________________");                       \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = ((a[i]) ^ (b[i]));                                                           \
        }                                                                                       \
    }

generate_kernel_MPIR_LBXOR(mpir_typename_int, int);
generate_kernel_MPIR_LBXOR(mpir_typename_long, long);
generate_kernel_MPIR_LBXOR(mpir_typename_short, short);
generate_kernel_MPIR_LBXOR(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPIR_LBXOR(mpir_typename_unsigned, unsigned);
generate_kernel_MPIR_LBXOR(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPIR_LBXOR(mpir_typename_long_long, long long);
generate_kernel_MPIR_LBXOR(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPIR_LBXOR(mpir_typename_signed_char, signed char);
generate_kernel_MPIR_LBXOR(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPIR_LBXOR(mpir_typename_int8_t, int8_t);
generate_kernel_MPIR_LBXOR(mpir_typename_int16_t, int16_t);
generate_kernel_MPIR_LBXOR(mpir_typename_int32_t, int32_t);
generate_kernel_MPIR_LBXOR(mpir_typename_int64_t, int64_t);
generate_kernel_MPIR_LBXOR(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPIR_LBXOR(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPIR_LBXOR(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPIR_LBXOR(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPIR_LBXOR(mpir_typename_char, char);
generate_kernel_MPIR_LBXOR(mpir_typename_integer, MPI_Fint);
generate_kernel_MPIR_LBXOR(mpir_typename_aint, MPI_Aint);
generate_kernel_MPIR_LBXOR(mpir_typename_offset, MPI_Offset);
generate_kernel_MPIR_LBXOR(mpir_typename_count, MPI_Count);
generate_kernel_MPIR_LBXOR(mpir_typename_character, char);
generate_kernel_MPIR_LBXOR(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPIR_LBXOR(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPIR_LBXOR(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPIR_LBXOR(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPIR_LBXOR(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPIR_LBXOR(mpir_typename_byte, unsigned char);

#define generate_kernel_MPIR_LLAND(type_name_, c_type_)                                          \
    __global__ void kernel_##type_name_##_##MPIR_LLAND(c_type_ *a, c_type_ *b, int n)            \
    {                                                                                           \
        printf("_______________Inside LLAND Kernel____________________");                       \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = ((a[i]) && (b[i]));                                                          \
        }                                                                                       \
    }

generate_kernel_MPIR_LLAND(mpir_typename_int, int);
generate_kernel_MPIR_LLAND(mpir_typename_long, long);
generate_kernel_MPIR_LLAND(mpir_typename_short, short);
generate_kernel_MPIR_LLAND(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPIR_LLAND(mpir_typename_unsigned, unsigned);
generate_kernel_MPIR_LLAND(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPIR_LLAND(mpir_typename_long_long, long long);
generate_kernel_MPIR_LLAND(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPIR_LLAND(mpir_typename_signed_char, signed char);
generate_kernel_MPIR_LLAND(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPIR_LLAND(mpir_typename_int8_t, int8_t);
generate_kernel_MPIR_LLAND(mpir_typename_int16_t, int16_t);
generate_kernel_MPIR_LLAND(mpir_typename_int32_t, int32_t);
generate_kernel_MPIR_LLAND(mpir_typename_int64_t, int64_t);
generate_kernel_MPIR_LLAND(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPIR_LLAND(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPIR_LLAND(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPIR_LLAND(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPIR_LLAND(mpir_typename_char, char);
generate_kernel_MPIR_LLAND(mpir_typename_integer, MPI_Fint);
generate_kernel_MPIR_LLAND(mpir_typename_aint, MPI_Aint);
generate_kernel_MPIR_LLAND(mpir_typename_offset, MPI_Offset);
generate_kernel_MPIR_LLAND(mpir_typename_count, MPI_Count);
generate_kernel_MPIR_LLAND(mpir_typename_character, char);
generate_kernel_MPIR_LLAND(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_float, float);
generate_kernel_MPIR_LLAND(mpir_typename_double, double);
generate_kernel_MPIR_LLAND(mpir_typename_real, MPIR_FC_REAL_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_long_double, long double);
generate_kernel_MPIR_LLAND(mpir_typename_real4, MPIR_REAL4_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_real8, MPIR_REAL8_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_real16, MPIR_REAL16_CTYPE);
generate_kernel_MPIR_LLAND(mpir_typename_float16, _Float16);
generate_kernel_MPIR_LLAND(mpir_typename_logical, MPI_Fint);
generate_kernel_MPIR_LLAND(mpir_typename_c_bool, _Bool);
generate_kernel_MPIR_LLAND(mpir_typename_cxx_bool_value, MPIR_CXX_BOOL_CTYPE);

#define generate_kernel_MPIR_LLOR(type_name_, c_type_)                                           \
    __global__ void kernel_##type_name_##_##MPIR_LLOR(c_type_ *a, c_type_ *b, int n)             \
    {                                                                                           \
        printf("_______________Inside LLOR Kernel____________________");                        \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = ((a[i]) || (b[i]));                                                          \
        }                                                                                       \
    }

generate_kernel_MPIR_LLOR(mpir_typename_int, int);
generate_kernel_MPIR_LLOR(mpir_typename_long, long);
generate_kernel_MPIR_LLOR(mpir_typename_short, short);
generate_kernel_MPIR_LLOR(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPIR_LLOR(mpir_typename_unsigned, unsigned);
generate_kernel_MPIR_LLOR(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPIR_LLOR(mpir_typename_long_long, long long);
generate_kernel_MPIR_LLOR(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPIR_LLOR(mpir_typename_signed_char, signed char);
generate_kernel_MPIR_LLOR(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPIR_LLOR(mpir_typename_int8_t, int8_t);
generate_kernel_MPIR_LLOR(mpir_typename_int16_t, int16_t);
generate_kernel_MPIR_LLOR(mpir_typename_int32_t, int32_t);
generate_kernel_MPIR_LLOR(mpir_typename_int64_t, int64_t);
generate_kernel_MPIR_LLOR(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPIR_LLOR(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPIR_LLOR(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPIR_LLOR(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPIR_LLOR(mpir_typename_char, char);
generate_kernel_MPIR_LLOR(mpir_typename_integer, MPI_Fint);
generate_kernel_MPIR_LLOR(mpir_typename_aint, MPI_Aint);
generate_kernel_MPIR_LLOR(mpir_typename_offset, MPI_Offset);
generate_kernel_MPIR_LLOR(mpir_typename_count, MPI_Count);
generate_kernel_MPIR_LLOR(mpir_typename_character, char);
generate_kernel_MPIR_LLOR(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_float, float);
generate_kernel_MPIR_LLOR(mpir_typename_double, double);
generate_kernel_MPIR_LLOR(mpir_typename_real, MPIR_FC_REAL_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_long_double, long double);
generate_kernel_MPIR_LLOR(mpir_typename_real4, MPIR_REAL4_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_real8, MPIR_REAL8_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_real16, MPIR_REAL16_CTYPE);
generate_kernel_MPIR_LLOR(mpir_typename_float16, _Float16);
generate_kernel_MPIR_LLOR(mpir_typename_logical, MPI_Fint);
generate_kernel_MPIR_LLOR(mpir_typename_c_bool, _Bool);
generate_kernel_MPIR_LLOR(mpir_typename_cxx_bool_value, MPIR_CXX_BOOL_CTYPE);

#define generate_kernel_MPIR_LLXOR(type_name_, c_type_)                                          \
    __global__ void kernel_##type_name_##_##MPIR_LLXOR(c_type_ *a, c_type_ *b, int n)            \
    {                                                                                           \
        printf("_______________Inside LLXOR Kernel____________________");                       \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = (((a[i]) && (!b[i])) || ((!a[i]) && (b[i])));                                \
        }                                                                                       \
    }

generate_kernel_MPIR_LLXOR(mpir_typename_int, int);
generate_kernel_MPIR_LLXOR(mpir_typename_long, long);
generate_kernel_MPIR_LLXOR(mpir_typename_short, short);
generate_kernel_MPIR_LLXOR(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPIR_LLXOR(mpir_typename_unsigned, unsigned);
generate_kernel_MPIR_LLXOR(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPIR_LLXOR(mpir_typename_long_long, long long);
generate_kernel_MPIR_LLXOR(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPIR_LLXOR(mpir_typename_signed_char, signed char);
generate_kernel_MPIR_LLXOR(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPIR_LLXOR(mpir_typename_int8_t, int8_t);
generate_kernel_MPIR_LLXOR(mpir_typename_int16_t, int16_t);
generate_kernel_MPIR_LLXOR(mpir_typename_int32_t, int32_t);
generate_kernel_MPIR_LLXOR(mpir_typename_int64_t, int64_t);
generate_kernel_MPIR_LLXOR(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPIR_LLXOR(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPIR_LLXOR(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPIR_LLXOR(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPIR_LLXOR(mpir_typename_char, char);
generate_kernel_MPIR_LLXOR(mpir_typename_integer, MPI_Fint);
generate_kernel_MPIR_LLXOR(mpir_typename_aint, MPI_Aint);
generate_kernel_MPIR_LLXOR(mpir_typename_offset, MPI_Offset);
generate_kernel_MPIR_LLXOR(mpir_typename_count, MPI_Count);
generate_kernel_MPIR_LLXOR(mpir_typename_character, char);
generate_kernel_MPIR_LLXOR(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_float, float);
generate_kernel_MPIR_LLXOR(mpir_typename_double, double);
generate_kernel_MPIR_LLXOR(mpir_typename_real, MPIR_FC_REAL_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_long_double, long double);
generate_kernel_MPIR_LLXOR(mpir_typename_real4, MPIR_REAL4_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_real8, MPIR_REAL8_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_real16, MPIR_REAL16_CTYPE);
generate_kernel_MPIR_LLXOR(mpir_typename_float16, _Float16);
generate_kernel_MPIR_LLXOR(mpir_typename_logical, MPI_Fint);
generate_kernel_MPIR_LLXOR(mpir_typename_c_bool, _Bool);
generate_kernel_MPIR_LLXOR(mpir_typename_cxx_bool_value, MPIR_CXX_BOOL_CTYPE);

#define generate_kernel_MPIR_LPROD(type_name_, c_type_)                                          \
    __global__ void kernel_##type_name_##_##MPIR_LPROD(c_type_ *a, c_type_ *b, int n)            \
    {                                                                                           \
        printf("_______________Inside LPROD Kernel____________________");                       \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = a[i] * b[i];                                                                 \
        }                                                                                       \
    }

generate_kernel_MPIR_LPROD(mpir_typename_int, int);
generate_kernel_MPIR_LPROD(mpir_typename_long, long);
generate_kernel_MPIR_LPROD(mpir_typename_short, short);
generate_kernel_MPIR_LPROD(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPIR_LPROD(mpir_typename_unsigned, unsigned);
generate_kernel_MPIR_LPROD(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPIR_LPROD(mpir_typename_long_long, long long);
generate_kernel_MPIR_LPROD(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPIR_LPROD(mpir_typename_signed_char, signed char);
generate_kernel_MPIR_LPROD(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPIR_LPROD(mpir_typename_int8_t, int8_t);
generate_kernel_MPIR_LPROD(mpir_typename_int16_t, int16_t);
generate_kernel_MPIR_LPROD(mpir_typename_int32_t, int32_t);
generate_kernel_MPIR_LPROD(mpir_typename_int64_t, int64_t);
generate_kernel_MPIR_LPROD(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPIR_LPROD(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPIR_LPROD(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPIR_LPROD(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPIR_LPROD(mpir_typename_char, char);
generate_kernel_MPIR_LPROD(mpir_typename_integer, MPI_Fint);
generate_kernel_MPIR_LPROD(mpir_typename_aint, MPI_Aint);
generate_kernel_MPIR_LPROD(mpir_typename_offset, MPI_Offset);
generate_kernel_MPIR_LPROD(mpir_typename_count, MPI_Count);
generate_kernel_MPIR_LPROD(mpir_typename_character, char);
generate_kernel_MPIR_LPROD(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_float, float);
generate_kernel_MPIR_LPROD(mpir_typename_double, double);
generate_kernel_MPIR_LPROD(mpir_typename_real, MPIR_FC_REAL_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_long_double, long double);
generate_kernel_MPIR_LPROD(mpir_typename_real4, MPIR_REAL4_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_real8, MPIR_REAL8_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_real16, MPIR_REAL16_CTYPE);
generate_kernel_MPIR_LPROD(mpir_typename_float16, _Float16);
/*generate_kernel_MPIR_LPROD(mpir_typename_complex, s_fc_complex);
generate_kernel_MPIR_LPROD(mpir_typename_c_float_complex, float _Complex);
generate_kernel_MPIR_LPROD(mpir_typename_c_double_complex, double _Complex);
generate_kernel_MPIR_LPROD(mpir_typename_c_long_double_complex, long double _Complex);
generate_kernel_MPIR_LPROD(mpir_typename_double_complex, d_fc_complex);
generate_kernel_MPIR_LPROD(mpir_typename_complex8, s_complex);
generate_kernel_MPIR_LPROD(mpir_typename_complex16, d_complex);
generate_kernel_MPIR_LPROD(mpir_typename_cxx_complex_value, s_complex);
generate_kernel_MPIR_LPROD(mpir_typename_cxx_double_complex_value, d_complex);
generate_kernel_MPIR_LPROD(mpir_typename_cxx_long_double_complex_value, ld_complex);*/

#define generate_kernel_MPIR_LSUM(type_name_, c_type_)                                           \
    __global__ void kernel_##type_name_##_##MPIR_LSUM(c_type_ *a, c_type_ *b, int n)             \
    {                                                                                           \
        printf("_______________Inside LSUM Kernel____________________");                        \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = a[i] + b[i];                                                                 \
        }                                                                                       \
    }

generate_kernel_MPIR_LSUM(mpir_typename_int, int);
generate_kernel_MPIR_LSUM(mpir_typename_long, long);
generate_kernel_MPIR_LSUM(mpir_typename_short, short);
generate_kernel_MPIR_LSUM(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPIR_LSUM(mpir_typename_unsigned, unsigned);
generate_kernel_MPIR_LSUM(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPIR_LSUM(mpir_typename_long_long, long long);
generate_kernel_MPIR_LSUM(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPIR_LSUM(mpir_typename_signed_char, signed char);
generate_kernel_MPIR_LSUM(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPIR_LSUM(mpir_typename_int8_t, int8_t);
generate_kernel_MPIR_LSUM(mpir_typename_int16_t, int16_t);
generate_kernel_MPIR_LSUM(mpir_typename_int32_t, int32_t);
generate_kernel_MPIR_LSUM(mpir_typename_int64_t, int64_t);
generate_kernel_MPIR_LSUM(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPIR_LSUM(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPIR_LSUM(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPIR_LSUM(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPIR_LSUM(mpir_typename_char, char);
generate_kernel_MPIR_LSUM(mpir_typename_integer, MPI_Fint);
generate_kernel_MPIR_LSUM(mpir_typename_aint, MPI_Aint);
generate_kernel_MPIR_LSUM(mpir_typename_offset, MPI_Offset);
generate_kernel_MPIR_LSUM(mpir_typename_count, MPI_Count);
generate_kernel_MPIR_LSUM(mpir_typename_character, char);
generate_kernel_MPIR_LSUM(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_float, float);
generate_kernel_MPIR_LSUM(mpir_typename_double, double);
generate_kernel_MPIR_LSUM(mpir_typename_real, MPIR_FC_REAL_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_long_double, long double);
generate_kernel_MPIR_LSUM(mpir_typename_real4, MPIR_REAL4_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_real8, MPIR_REAL8_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_real16, MPIR_REAL16_CTYPE);
generate_kernel_MPIR_LSUM(mpir_typename_float16, _Float16);
/*generate_kernel_MPIR_LSUM(mpir_typename_complex, s_fc_complex);
generate_kernel_MPIR_LSUM(mpir_typename_c_float_complex, float _Complex);
generate_kernel_MPIR_LSUM(mpir_typename_c_double_complex, double _Complex);
generate_kernel_MPIR_LSUM(mpir_typename_c_long_double_complex, long double _Complex);
generate_kernel_MPIR_LSUM(mpir_typename_double_complex, d_fc_complex);
generate_kernel_MPIR_LSUM(mpir_typename_complex8, s_complex);
generate_kernel_MPIR_LSUM(mpir_typename_complex16, d_complex);
generate_kernel_MPIR_LSUM(mpir_typename_cxx_complex_value, s_complex);
generate_kernel_MPIR_LSUM(mpir_typename_cxx_double_complex_value, d_complex);
generate_kernel_MPIR_LSUM(mpir_typename_cxx_long_double_complex_value, ld_complex);*/

#define generate_kernel_MPL_MAX(type_name_, c_type_)                                             \
    __global__ void kernel_##type_name_##_##MPL_MAX(c_type_ *a, c_type_ *b, int n)               \
    {                                                                                           \
        printf("_______________Inside MPL_MAX Kernel____________________");                     \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = (((a[i]) > (b[i])) ? (a[i]) : (b[i]));                                       \
        }                                                                                       \
    }

generate_kernel_MPL_MAX(mpir_typename_int, int);
generate_kernel_MPL_MAX(mpir_typename_long, long);
generate_kernel_MPL_MAX(mpir_typename_short, short);
generate_kernel_MPL_MAX(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPL_MAX(mpir_typename_unsigned, unsigned);
generate_kernel_MPL_MAX(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPL_MAX(mpir_typename_long_long, long long);
generate_kernel_MPL_MAX(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPL_MAX(mpir_typename_signed_char, signed char);
generate_kernel_MPL_MAX(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPL_MAX(mpir_typename_int8_t, int8_t);
generate_kernel_MPL_MAX(mpir_typename_int16_t, int16_t);
generate_kernel_MPL_MAX(mpir_typename_int32_t, int32_t);
generate_kernel_MPL_MAX(mpir_typename_int64_t, int64_t);
generate_kernel_MPL_MAX(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPL_MAX(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPL_MAX(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPL_MAX(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPL_MAX(mpir_typename_char, char);
generate_kernel_MPL_MAX(mpir_typename_integer, MPI_Fint);
generate_kernel_MPL_MAX(mpir_typename_aint, MPI_Aint);
generate_kernel_MPL_MAX(mpir_typename_offset, MPI_Offset);
generate_kernel_MPL_MAX(mpir_typename_count, MPI_Count);
generate_kernel_MPL_MAX(mpir_typename_character, char);
generate_kernel_MPL_MAX(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_float, float);
generate_kernel_MPL_MAX(mpir_typename_double, double);
generate_kernel_MPL_MAX(mpir_typename_real, MPIR_FC_REAL_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_long_double, long double);
generate_kernel_MPL_MAX(mpir_typename_real4, MPIR_REAL4_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_real8, MPIR_REAL8_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_real16, MPIR_REAL16_CTYPE);
generate_kernel_MPL_MAX(mpir_typename_float16, _Float16);

#define generate_kernel_MPL_MIN(type_name_, c_type_)                                             \
    __global__ void kernel_##type_name_##_##MPL_MIN(c_type_ *a, c_type_ *b, int n)               \
    {                                                                                           \
        printf("_______________Inside MPL_MIN Kernel____________________");                     \
        int i = threadIdx.x;                                          \
        if (i < n)                                                                              \
        {                                                                                       \
            a[i] = (((a[i]) < (b[i])) ? (a[i]) : (b[i]));                                       \
        }                                                                                       \
    }

generate_kernel_MPL_MIN(mpir_typename_int, int);
generate_kernel_MPL_MIN(mpir_typename_long, long);
generate_kernel_MPL_MIN(mpir_typename_short, short);
generate_kernel_MPL_MIN(mpir_typename_unsigned_short, unsigned short);
generate_kernel_MPL_MIN(mpir_typename_unsigned, unsigned);
generate_kernel_MPL_MIN(mpir_typename_unsigned_long, unsigned long);
generate_kernel_MPL_MIN(mpir_typename_long_long, long long);
generate_kernel_MPL_MIN(mpir_typename_unsigned_long_long, unsigned long long);
generate_kernel_MPL_MIN(mpir_typename_signed_char, signed char);
generate_kernel_MPL_MIN(mpir_typename_unsigned_char, unsigned char);
generate_kernel_MPL_MIN(mpir_typename_int8_t, int8_t);
generate_kernel_MPL_MIN(mpir_typename_int16_t, int16_t);
generate_kernel_MPL_MIN(mpir_typename_int32_t, int32_t);
generate_kernel_MPL_MIN(mpir_typename_int64_t, int64_t);
generate_kernel_MPL_MIN(mpir_typename_uint8_t, uint8_t);
generate_kernel_MPL_MIN(mpir_typename_uint16_t, uint16_t);
generate_kernel_MPL_MIN(mpir_typename_uint32_t, uint32_t);
generate_kernel_MPL_MIN(mpir_typename_uint64_t, uint64_t);
generate_kernel_MPL_MIN(mpir_typename_char, char);
generate_kernel_MPL_MIN(mpir_typename_integer, MPI_Fint);
generate_kernel_MPL_MIN(mpir_typename_aint, MPI_Aint);
generate_kernel_MPL_MIN(mpir_typename_offset, MPI_Offset);
generate_kernel_MPL_MIN(mpir_typename_count, MPI_Count);
generate_kernel_MPL_MIN(mpir_typename_character, char);
generate_kernel_MPL_MIN(mpir_typename_integer1, MPIR_INTEGER1_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_integer2, MPIR_INTEGER2_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_integer4, MPIR_INTEGER4_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_integer8, MPIR_INTEGER8_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_integer16, MPIR_INTEGER16_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_float, float);
generate_kernel_MPL_MIN(mpir_typename_double, double);
generate_kernel_MPL_MIN(mpir_typename_real, MPIR_FC_REAL_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_long_double, long double);
generate_kernel_MPL_MIN(mpir_typename_real4, MPIR_REAL4_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_real8, MPIR_REAL8_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_real16, MPIR_REAL16_CTYPE);
generate_kernel_MPL_MIN(mpir_typename_float16, _Float16);

#define generate_kernel_call(type_name_, c_type_, op_macro_)                                     \
    void call_##type_name_##_##op_macro_(c_type_ *a, c_type_ *b, int len)                        \
    {                                                                                           \
        kernel_##type_name_##_##op_macro_<<<1, len>>>(a, b, len);                                  \
    }                                                                       



generate_kernel_call(mpir_typename_int, int, MPIR_LBAND);
generate_kernel_call(mpir_typename_int, int, MPIR_LBOR);
generate_kernel_call(mpir_typename_int, int, MPIR_LBXOR);
generate_kernel_call(mpir_typename_int, int, MPIR_LLAND);
generate_kernel_call(mpir_typename_int, int, MPIR_LLOR);
generate_kernel_call(mpir_typename_int, int, MPIR_LLXOR);
generate_kernel_call(mpir_typename_int, int, MPIR_LPROD);
generate_kernel_call(mpir_typename_int, int, MPIR_LSUM);
generate_kernel_call(mpir_typename_int, int, MPL_MAX);
generate_kernel_call(mpir_typename_int, int, MPL_MIN);

generate_kernel_call(mpir_typename_long, long, MPIR_LBAND);
generate_kernel_call(mpir_typename_long, long, MPIR_LBOR);
generate_kernel_call(mpir_typename_long, long, MPIR_LBXOR);
generate_kernel_call(mpir_typename_long, long, MPIR_LLAND);
generate_kernel_call(mpir_typename_long, long, MPIR_LLOR);
generate_kernel_call(mpir_typename_long, long, MPIR_LLXOR);
generate_kernel_call(mpir_typename_long, long, MPIR_LPROD);
generate_kernel_call(mpir_typename_long, long, MPIR_LSUM);
generate_kernel_call(mpir_typename_long, long, MPL_MAX);
generate_kernel_call(mpir_typename_long, long, MPL_MIN);

generate_kernel_call(mpir_typename_short, short, MPIR_LBAND);
generate_kernel_call(mpir_typename_short, short, MPIR_LBOR);
generate_kernel_call(mpir_typename_short, short, MPIR_LBXOR);
generate_kernel_call(mpir_typename_short, short, MPIR_LLAND);
generate_kernel_call(mpir_typename_short, short, MPIR_LLOR);
generate_kernel_call(mpir_typename_short, short, MPIR_LLXOR);
generate_kernel_call(mpir_typename_short, short, MPIR_LPROD);
generate_kernel_call(mpir_typename_short, short, MPIR_LSUM);
generate_kernel_call(mpir_typename_short, short, MPL_MAX);
generate_kernel_call(mpir_typename_short, short, MPL_MIN);

generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPIR_LBAND);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPIR_LBOR);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPIR_LBXOR);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPIR_LLAND);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPIR_LLOR);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPIR_LLXOR);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPIR_LPROD);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPIR_LSUM);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPL_MAX);
generate_kernel_call(mpir_typename_unsigned_short, unsigned short, MPL_MIN);

generate_kernel_call(mpir_typename_unsigned, unsigned, MPIR_LBAND);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPIR_LBOR);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPIR_LBXOR);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPIR_LLAND);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPIR_LLOR);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPIR_LLXOR);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPIR_LPROD);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPIR_LSUM);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPL_MAX);
generate_kernel_call(mpir_typename_unsigned, unsigned, MPL_MIN);

generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPIR_LBAND);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPIR_LBOR);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPIR_LBXOR);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPIR_LLAND);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPIR_LLOR);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPIR_LLXOR);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPIR_LPROD);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPIR_LSUM);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPL_MAX);
generate_kernel_call(mpir_typename_unsigned_long, unsigned long, MPL_MIN);

generate_kernel_call(mpir_typename_long_long, long long, MPIR_LBAND);
generate_kernel_call(mpir_typename_long_long, long long, MPIR_LBOR);
generate_kernel_call(mpir_typename_long_long, long long, MPIR_LBXOR);
generate_kernel_call(mpir_typename_long_long, long long, MPIR_LLAND);
generate_kernel_call(mpir_typename_long_long, long long, MPIR_LLOR);
generate_kernel_call(mpir_typename_long_long, long long, MPIR_LLXOR);
generate_kernel_call(mpir_typename_long_long, long long, MPIR_LPROD);
generate_kernel_call(mpir_typename_long_long, long long, MPIR_LSUM);
generate_kernel_call(mpir_typename_long_long, long long, MPL_MAX);
generate_kernel_call(mpir_typename_long_long, long long, MPL_MIN);

generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPIR_LBAND);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPIR_LBOR);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPIR_LBXOR);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPIR_LLAND);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPIR_LLOR);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPIR_LLXOR);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPIR_LPROD);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPIR_LSUM);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPL_MAX);
generate_kernel_call(mpir_typename_unsigned_long_long, unsigned long long, MPL_MIN);

generate_kernel_call(mpir_typename_signed_char, signed char, MPIR_LBAND);
generate_kernel_call(mpir_typename_signed_char, signed char, MPIR_LBOR);
generate_kernel_call(mpir_typename_signed_char, signed char, MPIR_LBXOR);
generate_kernel_call(mpir_typename_signed_char, signed char, MPIR_LLAND);
generate_kernel_call(mpir_typename_signed_char, signed char, MPIR_LLOR);
generate_kernel_call(mpir_typename_signed_char, signed char, MPIR_LLXOR);
generate_kernel_call(mpir_typename_signed_char, signed char, MPIR_LPROD);
generate_kernel_call(mpir_typename_signed_char, signed char, MPIR_LSUM);
generate_kernel_call(mpir_typename_signed_char, signed char, MPL_MAX);
generate_kernel_call(mpir_typename_signed_char, signed char, MPL_MIN);

generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPIR_LBAND);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPIR_LBOR);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPIR_LBXOR);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPIR_LLAND);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPIR_LLOR);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPIR_LLXOR);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPIR_LPROD);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPIR_LSUM);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPL_MAX);
generate_kernel_call(mpir_typename_unsigned_char, unsigned char, MPL_MIN);

generate_kernel_call(mpir_typename_int8_t, int8_t, MPIR_LBAND);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPIR_LBOR);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPIR_LBXOR);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPIR_LLAND);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPIR_LLOR);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPIR_LLXOR);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPIR_LPROD);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPIR_LSUM);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPL_MAX);
generate_kernel_call(mpir_typename_int8_t, int8_t, MPL_MIN);

generate_kernel_call(mpir_typename_int16_t, int16_t, MPIR_LBAND);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPIR_LBOR);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPIR_LBXOR);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPIR_LLAND);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPIR_LLOR);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPIR_LLXOR);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPIR_LPROD);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPIR_LSUM);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPL_MAX);
generate_kernel_call(mpir_typename_int16_t, int16_t, MPL_MIN);

generate_kernel_call(mpir_typename_int32_t, int32_t, MPIR_LBAND);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPIR_LBOR);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPIR_LBXOR);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPIR_LLAND);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPIR_LLOR);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPIR_LLXOR);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPIR_LPROD);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPIR_LSUM);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPL_MAX);
generate_kernel_call(mpir_typename_int32_t, int32_t, MPL_MIN);

generate_kernel_call(mpir_typename_int64_t, int64_t, MPIR_LBAND);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPIR_LBOR);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPIR_LBXOR);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPIR_LLAND);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPIR_LLOR);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPIR_LLXOR);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPIR_LPROD);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPIR_LSUM);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPL_MAX);
generate_kernel_call(mpir_typename_int64_t, int64_t, MPL_MIN);

generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPIR_LBAND);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPIR_LBOR);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPIR_LBXOR);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPIR_LLAND);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPIR_LLOR);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPIR_LLXOR);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPIR_LPROD);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPIR_LSUM);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPL_MAX);
generate_kernel_call(mpir_typename_uint8_t, uint8_t, MPL_MIN);

generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPIR_LBAND);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPIR_LBOR);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPIR_LBXOR);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPIR_LLAND);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPIR_LLOR);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPIR_LLXOR);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPIR_LPROD);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPIR_LSUM);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPL_MAX);
generate_kernel_call(mpir_typename_uint16_t, uint16_t, MPL_MIN);

generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPIR_LBAND);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPIR_LBOR);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPIR_LBXOR);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPIR_LLAND);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPIR_LLOR);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPIR_LLXOR);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPIR_LPROD);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPIR_LSUM);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPL_MAX);
generate_kernel_call(mpir_typename_uint32_t, uint32_t, MPL_MIN);

generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPIR_LBAND);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPIR_LBOR);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPIR_LBXOR);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPIR_LLAND);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPIR_LLOR);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPIR_LLXOR);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPIR_LPROD);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPIR_LSUM);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPL_MAX);
generate_kernel_call(mpir_typename_uint64_t, uint64_t, MPL_MIN);

generate_kernel_call(mpir_typename_char, char, MPIR_LBAND);
generate_kernel_call(mpir_typename_char, char, MPIR_LBOR);
generate_kernel_call(mpir_typename_char, char, MPIR_LBXOR);
generate_kernel_call(mpir_typename_char, char, MPIR_LLAND);
generate_kernel_call(mpir_typename_char, char, MPIR_LLOR);
generate_kernel_call(mpir_typename_char, char, MPIR_LLXOR);
generate_kernel_call(mpir_typename_char, char, MPIR_LPROD);
generate_kernel_call(mpir_typename_char, char, MPIR_LSUM);
generate_kernel_call(mpir_typename_char, char, MPL_MAX);
generate_kernel_call(mpir_typename_char, char, MPL_MIN);

generate_kernel_call(mpir_typename_integer, MPI_Fint, MPIR_LBAND);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPIR_LBOR);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPIR_LBXOR);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPIR_LLAND);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPIR_LLOR);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPIR_LLXOR);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPIR_LPROD);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPIR_LSUM);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPL_MAX);
generate_kernel_call(mpir_typename_integer, MPI_Fint, MPL_MIN);

generate_kernel_call(mpir_typename_aint, MPI_Aint, MPIR_LBAND);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPIR_LBOR);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPIR_LBXOR);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPIR_LLAND);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPIR_LLOR);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPIR_LLXOR);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPIR_LPROD);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPIR_LSUM);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPL_MAX);
generate_kernel_call(mpir_typename_aint, MPI_Aint, MPL_MIN);

generate_kernel_call(mpir_typename_offset, MPI_Offset, MPIR_LBAND);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPIR_LBOR);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPIR_LBXOR);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPIR_LLAND);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPIR_LLOR);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPIR_LLXOR);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPIR_LPROD);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPIR_LSUM);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPL_MAX);
generate_kernel_call(mpir_typename_offset, MPI_Offset, MPL_MIN);

generate_kernel_call(mpir_typename_count, MPI_Count, MPIR_LBAND);
generate_kernel_call(mpir_typename_count, MPI_Count, MPIR_LBOR);
generate_kernel_call(mpir_typename_count, MPI_Count, MPIR_LBXOR);
generate_kernel_call(mpir_typename_count, MPI_Count, MPIR_LLAND);
generate_kernel_call(mpir_typename_count, MPI_Count, MPIR_LLOR);
generate_kernel_call(mpir_typename_count, MPI_Count, MPIR_LLXOR);
generate_kernel_call(mpir_typename_count, MPI_Count, MPIR_LPROD);
generate_kernel_call(mpir_typename_count, MPI_Count, MPIR_LSUM);
generate_kernel_call(mpir_typename_count, MPI_Count, MPL_MAX);
generate_kernel_call(mpir_typename_count, MPI_Count, MPL_MIN);

generate_kernel_call(mpir_typename_character, char, MPIR_LBAND);
generate_kernel_call(mpir_typename_character, char, MPIR_LBOR);
generate_kernel_call(mpir_typename_character, char, MPIR_LBXOR);
generate_kernel_call(mpir_typename_character, char, MPIR_LLAND);
generate_kernel_call(mpir_typename_character, char, MPIR_LLOR);
generate_kernel_call(mpir_typename_character, char, MPIR_LLXOR);
generate_kernel_call(mpir_typename_character, char, MPIR_LPROD);
generate_kernel_call(mpir_typename_character, char, MPIR_LSUM);
generate_kernel_call(mpir_typename_character, char, MPL_MAX);
generate_kernel_call(mpir_typename_character, char, MPL_MIN);

generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPIR_LBAND);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPIR_LBOR);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPIR_LBXOR);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_integer1, MPIR_INTEGER1_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPIR_LBAND);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPIR_LBOR);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPIR_LBXOR);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_integer2, MPIR_INTEGER2_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPIR_LBAND);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPIR_LBOR);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPIR_LBXOR);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_integer4, MPIR_INTEGER4_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPIR_LBAND);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPIR_LBOR);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPIR_LBXOR);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_integer8, MPIR_INTEGER8_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPIR_LBAND);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPIR_LBOR);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPIR_LBXOR);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_integer16, MPIR_INTEGER16_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_float, float, MPIR_LLAND);
generate_kernel_call(mpir_typename_float, float, MPIR_LLOR);
generate_kernel_call(mpir_typename_float, float, MPIR_LLXOR);
generate_kernel_call(mpir_typename_float, float, MPIR_LPROD);
generate_kernel_call(mpir_typename_float, float, MPIR_LSUM);
generate_kernel_call(mpir_typename_float, float, MPL_MAX);
generate_kernel_call(mpir_typename_float, float, MPL_MIN);

generate_kernel_call(mpir_typename_double, double, MPIR_LLAND);
generate_kernel_call(mpir_typename_double, double, MPIR_LLOR);
generate_kernel_call(mpir_typename_double, double, MPIR_LLXOR);
generate_kernel_call(mpir_typename_double, double, MPIR_LPROD);
generate_kernel_call(mpir_typename_double, double, MPIR_LSUM);
generate_kernel_call(mpir_typename_double, double, MPL_MAX);
generate_kernel_call(mpir_typename_double, double, MPL_MIN);

generate_kernel_call(mpir_typename_real, MPIR_FC_REAL_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_real, MPIR_FC_REAL_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_real, MPIR_FC_REAL_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_real, MPIR_FC_REAL_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_real, MPIR_FC_REAL_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_real, MPIR_FC_REAL_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_real, MPIR_FC_REAL_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_double_precision, MPIR_FC_DOUBLE_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_long_double, long double, MPIR_LLAND);
generate_kernel_call(mpir_typename_long_double, long double, MPIR_LLOR);
generate_kernel_call(mpir_typename_long_double, long double, MPIR_LLXOR);
generate_kernel_call(mpir_typename_long_double, long double, MPIR_LPROD);
generate_kernel_call(mpir_typename_long_double, long double, MPIR_LSUM);
generate_kernel_call(mpir_typename_long_double, long double, MPL_MAX);
generate_kernel_call(mpir_typename_long_double, long double, MPL_MIN);

generate_kernel_call(mpir_typename_real4, MPIR_REAL4_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_real4, MPIR_REAL4_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_real4, MPIR_REAL4_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_real4, MPIR_REAL4_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_real4, MPIR_REAL4_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_real4, MPIR_REAL4_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_real4, MPIR_REAL4_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_real8, MPIR_REAL8_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_real8, MPIR_REAL8_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_real8, MPIR_REAL8_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_real8, MPIR_REAL8_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_real8, MPIR_REAL8_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_real8, MPIR_REAL8_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_real8, MPIR_REAL8_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_real16, MPIR_REAL16_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_real16, MPIR_REAL16_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_real16, MPIR_REAL16_CTYPE, MPIR_LLXOR);
generate_kernel_call(mpir_typename_real16, MPIR_REAL16_CTYPE, MPIR_LPROD);
generate_kernel_call(mpir_typename_real16, MPIR_REAL16_CTYPE, MPIR_LSUM);
generate_kernel_call(mpir_typename_real16, MPIR_REAL16_CTYPE, MPL_MAX);
generate_kernel_call(mpir_typename_real16, MPIR_REAL16_CTYPE, MPL_MIN);

generate_kernel_call(mpir_typename_float16, _Float16, MPIR_LLAND);
generate_kernel_call(mpir_typename_float16, _Float16, MPIR_LLOR);
generate_kernel_call(mpir_typename_float16, _Float16, MPIR_LLXOR);
generate_kernel_call(mpir_typename_float16, _Float16, MPIR_LPROD);
generate_kernel_call(mpir_typename_float16, _Float16, MPIR_LSUM);
generate_kernel_call(mpir_typename_float16, _Float16, MPL_MAX);
generate_kernel_call(mpir_typename_float16, _Float16, MPL_MIN);

generate_kernel_call(mpir_typename_logical, MPI_Fint, MPIR_LLAND);
generate_kernel_call(mpir_typename_logical, MPI_Fint, MPIR_LLOR);
generate_kernel_call(mpir_typename_logical, MPI_Fint, MPIR_LLXOR);

generate_kernel_call(mpir_typename_c_bool, _Bool, MPIR_LLAND);
generate_kernel_call(mpir_typename_c_bool, _Bool, MPIR_LLOR);
generate_kernel_call(mpir_typename_c_bool, _Bool, MPIR_LLXOR);

generate_kernel_call(mpir_typename_cxx_bool_value, MPIR_CXX_BOOL_CTYPE, MPIR_LLAND);
generate_kernel_call(mpir_typename_cxx_bool_value, MPIR_CXX_BOOL_CTYPE, MPIR_LLOR);
generate_kernel_call(mpir_typename_cxx_bool_value, MPIR_CXX_BOOL_CTYPE, MPIR_LLXOR);
/*
generate_kernel_call(mpir_typename_complex, s_fc_complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_complex, s_fc_complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_c_float_complex, float _Complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_c_float_complex, float _Complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_c_double_complex, double _Complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_c_double_complex, double _Complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_c_long_double_complex, long double _Complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_c_long_double_complex, long double _Complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_double_complex, d_fc_complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_double_complex, d_fc_complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_complex8, s_complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_complex8, s_complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_complex16, d_complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_complex16, d_complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_cxx_complex_value, s_complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_cxx_complex_value, s_complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_cxx_double_complex_value, d_complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_cxx_double_complex_value, d_complex, MPIR_LSUM);

generate_kernel_call(mpir_typename_cxx_long_double_complex_value, ld_complex, MPIR_LPROD);
generate_kernel_call(mpir_typename_cxx_long_double_complex_value, ld_complex, MPIR_LSUM);
*/
generate_kernel_call(mpir_typename_byte, unsigned char, MPIR_LBAND);
generate_kernel_call(mpir_typename_byte, unsigned char, MPIR_LBOR);
generate_kernel_call(mpir_typename_byte, unsigned char, MPIR_LBXOR);
}
