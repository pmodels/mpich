#include "hip/hip_runtime.h"
/* CC: nvcc -g */
/* lib_list: -lmpi */
/* run: mpirun -l -n 2 */

#include <mpi.h>
#include <stdio.h>
#include <assert.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
    int mpi_errno;
    int rank, size;
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    hipStream_t stream;
    hipStreamCreate(&stream);

    int N = 1000000;
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    if (rank == 0) {
        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
        }
    } else if (rank == 1) {
        for (int i = 0; i < N; i++) {
            y[i] = 2.0f;
        }
    }

    if (rank == 0) {
      #if 0  
        hipMemcpyAsync(d_x, x, N*sizeof(float), hipMemcpyHostToDevice, stream);

        mpi_errno = MPIX_Send_enqueue(d_x, N, MPI_FLOAT, 1, 0, MPI_COMM_WORLD, &stream);
      #else  
        mpi_errno = MPIX_Send_enqueue(x, N, MPI_FLOAT, 1, 0, MPI_COMM_WORLD, &stream);
      #endif  
        assert(mpi_errno == MPI_SUCCESS);

        hipStreamSynchronize(stream);
    } else if (rank == 1) {
        hipMemcpyAsync(d_y, y, N*sizeof(float), hipMemcpyHostToDevice, stream);

        mpi_errno = MPIX_Recv_enqueue(d_x, N, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE, &stream);
        assert(mpi_errno == MPI_SUCCESS);

        // Perform SAXPY on 1M elements
        saxpy<<<(N+255)/256, 256, 0, stream>>>(N, 2.0f, d_x, d_y);

        hipMemcpyAsync(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);
    }

    if (rank == 1) {
        float maxError = 0.0f;
        int errs = 0;
        for (int i = 0; i < N; i++) {
            if (abs(y[i] - 4.0f) > 0.01) {
                errs++;
                maxError = max(maxError, abs(y[i]-4.0f));
            }
        }
        if (errs > 0) {
            printf("%d errors, Max error: %f\n", errs, maxError);
        } else {
            printf("No Errors\n");
        }
    }

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    hipStreamDestroy(stream);
    MPI_Finalize();
}
