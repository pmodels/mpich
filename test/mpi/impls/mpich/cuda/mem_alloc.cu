#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <mpi.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(mpi_comm, call)                                  \
    {                                                               \
        const hipError_t error = call;                             \
        if (error != hipSuccess)                                   \
        {                                                           \
            fprintf(stderr, "An error occurred: \"%s\" at %s:%d\n", \
                    hipGetErrorString(error), __FILE__, __LINE__); \
            MPI_Abort(mpi_comm, error);                             \
        }                                                           \
    }

int main(int argc, char *argv[])
{
    int cuda_device_aware = 0;
    int cuda_managed_aware = 0;
    int len = 0, flag = 0;
    int *managed_buf = NULL;
    int *device_buf = NULL, *system_buf = NULL;
    int nranks = 0;
    int rank;
    MPI_Info info;
    MPI_Session session;
    MPI_Group wgroup;
    MPI_Comm system_comm;
    MPI_Comm cuda_managed_comm = MPI_COMM_NULL;
    MPI_Comm cuda_device_comm = MPI_COMM_NULL;

    // Usage mode: REQUESTED
    MPI_Info_create(&info);
    MPI_Info_set(info, "mpi_memory_alloc_kinds", "system,cuda:device,cuda:managed");
    MPI_Session_init(info, MPI_ERRORS_ARE_FATAL, &session);
    MPI_Info_free(&info);

    // Usage mode: PROVIDED
    MPI_Session_get_info(session, &info);
    MPI_Info_get_string(info, "mpi_memory_alloc_kinds", &len, NULL, &flag);

    if (flag) {
        char *val, *valptr, *kind;

        val = valptr = (char *) malloc(len);
        if (NULL == val)
            return 1;

        MPI_Info_get_string(info, "mpi_memory_alloc_kinds", &len, val, &flag);

        while ((kind = strsep(&val, ",")) != NULL) {
            if (strcasecmp(kind, "cuda:managed") == 0) {
                cuda_managed_aware = 1;
            } else if (strcasecmp(kind, "cuda:device") == 0) {
                cuda_device_aware = 1;
            }
        }
        free(valptr);
    }

    MPI_Info_free(&info);

    MPI_Group_from_session_pset(session, "mpi://WORLD", &wgroup);

    // Create a communicator for operations on system memory
    // Usage mode: ASSERTED
    MPI_Info_create(&info);
    MPI_Info_set(info, "mpi_assert_memory_alloc_kinds", "system");
    MPI_Comm_create_from_group(wgroup,
                               "org.mpi-forum.side-doc.mem-alloc-kind.cuda-example.system",
                               info, MPI_ERRORS_ABORT, &system_comm);
    MPI_Info_free(&info);

    MPI_Comm_size(system_comm, &nranks);
    MPI_Comm_rank(system_comm, &rank);

    /*** Check for CUDA awareness ***/

    // Note: MPI does not require homogeneous support
    // across all processes for memory allocation kinds.
    // This example chooses to use
    // CUDA managed allocations (or device allocations)
    // only when all processes report it is supported.

    // Check if all processes have CUDA managed support
    MPI_Allreduce(MPI_IN_PLACE, &cuda_managed_aware, 1, MPI_INT, MPI_LAND, system_comm);
    assert(cuda_managed_aware);

    // Create a communicator for operations that use
    // CUDA managed buffers.
    // Usage mode: ASSERTED
    MPI_Info_create(&info);
    MPI_Info_set(info, "mpi_assert_memory_alloc_kinds", "cuda:managed");
    MPI_Comm_create_from_group(wgroup,
                               "org.mpi-forum.side-doc.mem-alloc-kind.cuda-example.managed",
                               info, MPI_ERRORS_ABORT, &cuda_managed_comm);
    MPI_Info_free(&info);

    // Check if all processes have CUDA device support
    MPI_Allreduce(MPI_IN_PLACE, &cuda_device_aware, 1, MPI_INT, MPI_LAND, system_comm);
    assert(cuda_device_aware);
    // Create a communicator for operations that use
    // CUDA device buffers.
    // Usage mode: ASSERTED
    MPI_Info_create(&info);
    MPI_Info_set(info, "mpi_assert_memory_alloc_kinds", "cuda:device");
    MPI_Comm_create_from_group(wgroup,
                               "org.mpi-forum.side-doc.mem-alloc-kind.cuda-example.device",
                               info, MPI_ERRORS_ABORT, &cuda_device_comm);
    MPI_Info_free(&info);

    MPI_Group_free(&wgroup);

    /*** Execute using both types of memory ***/
    // Allocate managed buffer and initialize it
    CUDA_CHECK(system_comm,
               hipMallocManaged((void **) &managed_buf, sizeof(int), hipMemAttachGlobal));
    *managed_buf = 1;

    // Perform communication using cuda_managed_comm
    // if it's available.
    MPI_Allreduce(MPI_IN_PLACE, managed_buf, 1, MPI_INT, MPI_SUM, cuda_managed_comm);
    assert((*managed_buf) == nranks);

    CUDA_CHECK(system_comm, hipFree(managed_buf));

    // Allocate system buffer and initialize it
    // (using hipHostMalloc for better performance of hipMemcpy)
    CUDA_CHECK(system_comm, hipHostMalloc((void **) &system_buf, sizeof(int)));
    *system_buf = 1;

    // Allocate CUDA device buffer and initialize it
    CUDA_CHECK(system_comm, hipMalloc((void **) &device_buf, sizeof(int)));
    CUDA_CHECK(system_comm,
               hipMemcpyAsync(device_buf, system_buf, sizeof(int), hipMemcpyHostToDevice, 0));
    CUDA_CHECK(system_comm, hipStreamSynchronize(0));

    // Perform communication using cuda_device_comm
    // if it's available.
    MPI_Allreduce(MPI_IN_PLACE, device_buf, 1, MPI_INT, MPI_SUM, cuda_device_comm);
    CUDA_CHECK(system_comm,
               hipMemcpyAsync(system_buf, device_buf, sizeof(int), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK(system_comm, hipStreamSynchronize(0));
    assert((*system_buf) == nranks);

    if (cuda_managed_comm != MPI_COMM_NULL)
        MPI_Comm_disconnect(&cuda_managed_comm);
    if (cuda_device_comm != MPI_COMM_NULL)
        MPI_Comm_disconnect(&cuda_device_comm);
    MPI_Comm_disconnect(&system_comm);

    MPI_Session_finalize(&session);
    if (rank == 0) {
        printf("No Errors\n");
    }

    return 0;
}
